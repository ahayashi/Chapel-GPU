
#include <hip/hip_runtime.h>
#include <stdio.h>

#ifndef THREADS_PER_BLOCK
#define THREADS_PER_BLOCK 1024
#endif

__global__ void myKernel(int64_t *dA, size_t N) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < N) {
        dA[id] = dA[id] + 1;
    }
}

extern "C" {
    void kernel(int64_t *ptr, size_t N) {
        myKernel<<<ceil(((float)N)/THREADS_PER_BLOCK), THREADS_PER_BLOCK>>>(ptr, N);
    }
}
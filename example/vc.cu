
#include <hip/hip_runtime.h>
#ifndef THREADS_PER_BLOCK
#define THREADS_PER_BLOCK 1024
#endif

__global__ void vc(float *dA, float *dB, int N) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < N) {
        dA[id] = dB[id];
  }
}

extern "C" {
  void vcGPU(float* A, float *B, int start, int end, int GPUN) {
    float *dA, *dB;
    hipMalloc(&dA, sizeof(float) * GPUN);
    hipMalloc(&dB, sizeof(float) * GPUN);
    hipMemcpy(dB, B + start, sizeof(float) * GPUN, hipMemcpyHostToDevice);
    vc<<<ceil(((float)GPUN)/THREADS_PER_BLOCK), THREADS_PER_BLOCK>>>(dA, dB, GPUN);
    hipDeviceSynchronize();
    hipMemcpy(A + start, dA, sizeof(float) * GPUN, hipMemcpyDeviceToHost);
    hipFree(dA);
    hipFree(dB);
  }
}

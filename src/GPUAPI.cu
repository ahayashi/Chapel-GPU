
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <assert.h>
#include <hip/hip_runtime_api.h>

#define CUDA_ERROR_CHECK
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
  hipError_t err = hipGetLastError();
  if ( hipSuccess != err )
    {
      fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
               file, line, hipGetErrorString( err ) );
      exit( -1 );
    }

  // More careful checking. However, this will affect performance.
  // Comment away if needed.
  err = hipDeviceSynchronize();
  if( hipSuccess != err )
    {
      fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
               file, line, hipGetErrorString( err ) );
      exit( -1 );
    }
#endif
}

extern "C" {

  void GetDeviceCount(int *count) {
    CudaSafeCall(hipGetDeviceCount(count));
  }

  void GetDevice(int *device) {
    CudaSafeCall(hipGetDevice(device));
  }

  void SetDevice(int device) {
    CudaSafeCall(hipSetDevice(device));
  }

  void ProfilerStart() {
    CudaSafeCall(hipProfilerStart());
  }

  void ProfilerStop() {
    CudaSafeCall(hipProfilerStop());
  }

  void Malloc(void** devPtr, size_t size) {
    CudaSafeCall(hipMalloc(devPtr, size));
  }

  void Memcpy(void* dst, void* src, size_t count, int kind) {
      switch (kind) {
      case 0:
          CudaSafeCall(hipMemcpy(dst, src, count, hipMemcpyHostToDevice));
          break;
      case 1:
          CudaSafeCall(hipMemcpy(dst, src, count, hipMemcpyDeviceToHost));
          break;
      default:
          printf("Warning\n");
      }
  }
}
